#include "hip/hip_runtime.h"
// All rights reserved by
// Durga Keerthi Mandarapu, Vani Nagarajan, Artem Pelenitsyn, and Milind Kulkarni. 2024.
// Arkade: k-Nearest Neighbor Search With Non-Euclidean Distances using GPU Ray Tracing.

#include "DeviceCode.h"
#include "VarGlobal.h"
#include <optix_device.h>

using namespace owl;



// Variable global constante accedida desde el main
__constant__ EDA::GlobalVars optixLaunchParams;

// Template para soportar múltiples tipos de geometrías de esfera
template<typename SphereGeomType>

// ============================================================================
// Bounds Program - Se ejecuta en la GPU y calcula el AABB (Axis-Aligned Bounding Box)
// para una primitiva de tipo esfera.
// ============================================================================
inline __device__ void boundsProg(const void* geomData,
                                  box3f& primBounds,
                                  const int primID)
{
    // Cast del puntero genérico a nuestra estructura de geometría
    const SphereGeomType& self = *(const SphereGeomType*)geomData;

    // Obtenemos la esfera específica usando el ID de la primitiva
    const EDA::Point sphere = self.data_spheres[primID];

    // Creamos la bounding box (AABB) que encapsula exactamente la esfera.
    // Se extiende el box vacío en ambas direcciones a partir del centro:
    //  - primero hacia (centro - radio)
    //  - luego hacia (centro + radio)
    //
    // Esto genera una caja perfectamente ajustada a la esfera.
    primBounds = box3f()
            .extend(sphere.pt - self.rad)
            .extend(sphere.pt + self.rad);
}

// ============================================================================
// Crea el AABB exacto para cada esfera usando el programa de bounds de OptiX.
// Este macro enlaza tu función personalizada `boundsProg` con el sistema de BVH.
// ============================================================================
OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData, box3f &primBounds, const int primID){

    // geomData apunta al grupo de esferas (es un void* internamente)
    // primID: indice para acceder a la geometría que se está procesando en ese momento.
    boundsProg<EDA::SpheresGeom>(geomData,primBounds,primID);
}


// ============================================================================
// Intersección personalizada entre un rayo y una esfera.
// Este código se ejecuta cuando un rayo entra en una AABB y necesita decidir
// si realmente intersecta la geometría (esfera).
// En este MACRO se ejecuta el algoritmo de Filter & Refine para un Query Point.
// Acá se activa los RT Cores para la intersección de un Query RayGen y un AABB
// ============================================================================
OPTIX_INTERSECT_PROGRAM(Spheres)(){

    // ID de la primitiva con la que se está intentando intersectar en ese momento
    const int primID = optixGetPrimitiveIndex();

    // Obtenemos el conjunto de datos de la geometría actual
    const auto &self = owl::getProgramData<EDA::SpheresGeom>();

    // Extraemos la esfera correspondiente (ShaderCore)
    const EDA::Point sphere = self.data_spheres[primID];

    // Punto de origen del rayo en coordenadas del mundo (ShaderCore)
    const vec3f rayOrigin = optixGetWorldRayOrigin();

    // Extraer la norma a utilizar de las variables globales.
    int NORM = optixLaunchParams.NORM;

    // -----------------------------
    // Cálculo de distancia (L^p Norm)
    // -----------------------------
    float distance = 0.0;

    if (NORM == 0) { // Norm infinito (máxima componente absoluta)
        float dx = std::abs(sphere.pt.x - rayOrigin.x);
        float dy = std::abs(sphere.pt.y - rayOrigin.y);
        float dz = std::abs(sphere.pt.z - rayOrigin.z);
        distance = fmaxf(fmaxf(dx, dy), dz);
    }
    else if (NORM > 0) {
        // Norma L^p (sin raíz para eficiencia)
        distance = powf(std::abs(sphere.pt.x - rayOrigin.x), NORM) +
                   powf(std::abs(sphere.pt.y - rayOrigin.y), NORM) +
                   powf(std::abs(sphere.pt.z - rayOrigin.z), NORM);
    }

    // -----------------------------
    // Filtro de candidatos por radio (Filter)
    // Cálculo equivalente a sqrt(distance, NORM) < self.rad
    // -----------------------------
    if(distance < powf(self.rad, NORM)){

        // Accedemos al registro de vecinos actual
        auto& param = owl::getPRD<EDA::NeighKNN>();

        // Buscamos el vecino más lejano en la lista actual, el peor registrado.
        int max_idx = 0;
        for (int i = 1; i < KN; ++i) {
            if (param.res[i].dist > param.res[max_idx].dist) {
                max_idx = i;
            }
        }

        // Si la nueva distancia es mejor que el peor vecino, lo reemplazamos (Refine)
        if (distance < param.res[max_idx].dist) {
            param.res[max_idx].dist = distance;
            param.res[max_idx].idx  = primID;
        }
    }

}


// ============================================================================
// Programa de generación de rayos (rayGen).
// Se activa automáticamente en cada índice de lanzado al ejecutar owlLaunch2D().
// Este lanzador recorre los puntos de consulta (query points) y realiza trazado de rayos.
// ============================================================================
OPTIX_RAYGEN_PROGRAM(rayGen)(){

    // -----------------------------
    // 1. Obtener datos del programa
    // -----------------------------
    const auto& self = owl::getProgramData<EDA::RayGenData>();

    // Índice X de la RayGen Query lanzado (cada hilo procesa un punto de consulta)
    const int xID = optixGetLaunchIndex().x;

    // -----------------------------
    // 2. Inicializar los vecinos (KNN)
    // -----------------------------
    EDA::NeighKNN param{};
    for (auto & re : param.res) {
        re.idx = -1;
        re.dist = FLOAT_MAX;
    }

    // -----------------------------
    // 3. Construcción del rayo
    // -----------------------------
    // Se lanza un rayo desde el punto de consulta en dirección arbitraria (por ejemplo +Z)
    owl::Ray ray(self.query_pts[xID].pt, // Origen del rayo
                 vec3f(0,0,1), // Dirección (arbitraria en este contexto)
                 0, // Tipo del rayo
                 1.e-16f); // Para evitar auto intersecciones.

    // -----------------------------
    // 4. Trazado del rayo (intersección con la escena)
    // -----------------------------
    // Esto inicia el recorrido por el BVH y llama a OPTIX_INTERSECT_PROGRAM
    // param es modificado dentro del programa de intersección con los KNN encontrados
    owl::traceRay(self.world, ray, param);

    // -----------------------------
    // 5. Guardar resultados en el framebuffer global
    // En el Buffer de los resultados tiene num_search * KN de espacio reservado, para guardar por indices
    // los vecinos más cercanos de CADA query point.
    // -----------------------------
    for (int i = 0; i < KN; ++i) {

        // Cada k vecinos conseguidos le pertenece al query point correspondiente
        // El rango de índices en el que se ubica sus vecinos es: [ID*KN : ID*(KN+1)]
        int outputIndex = xID * KN + i;
        optixLaunchParams.frameBuffer[outputIndex].idx  = param.res[i].idx;
        optixLaunchParams.frameBuffer[outputIndex].dist = param.res[i].dist;
    }
}
