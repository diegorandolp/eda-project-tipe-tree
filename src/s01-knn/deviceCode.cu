#include "hip/hip_runtime.h"
// All rights reserved by
// Durga Keerthi Mandarapu, Vani Nagarajan, Artem Pelenitsyn, and Milind Kulkarni. 2024.
// Arkade: k-Nearest Neighbor Search With Non-Euclidean Distances using GPU Ray Tracing.

#include "deviceCode.h"
#include <optix_device.h>

using namespace owl;


// Acá agarramos la variable global owl creada en el MAIN.
__constant__ MyGlobals optixLaunchParams;

// bounding box programs
template<typename SphereGeomType>

// lo que esté con __device__ se ejecuta en la GPU

inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{
  const SphereGeomType &self = *(const SphereGeomType*)geomData;

  // rescatamos la esfera exacta
  const Sphere sphere = self.data_pts[primID];

    // Debemos crear el AABB apartir de la geometría para que ocupe exactamente el espacio que ocupa
    // box3f() crea una caja vacia, extend extiende CADA EJE DE LA ESFERA en - rad,
    // extenel 2do extend, extiende cada eje de la esfera ahora en + rad, cubriendo exactamente la esfera

    // tal que al excetender el box3f, está cubriendo exactamente la geometría.
  primBounds = box3f().extend(sphere.center - self.rad)
		                  .extend(sphere.center + self.rad);

}

// Si es que quiero usar una geometría personalizada, debo crear mi macro para que el AABB del BVH, sepa como englobal las geometrías
// acá está creando el AABB del esferas.
OPTIX_BOUNDS_PROGRAM(Spheres)(const void  *geomData,
  box3f       &primBounds,
  const int    primID)

    //geomData es spheresGroup, internamente se guarda como void, si se quiere acceder asus valores:
    //   const SphereGeomType &self = *(const SphereGeomType*)geomData;

    // primID: es el indice de la esfera la cual se está procesando y entrando a la función.
{ boundsProg<SpheresGeom>(geomData,primBounds,primID); }


// Este macro es el proceso de intersección de una geometría y un AABB.
// Corazón del recorrido e intersección personalizada con el BVH.

// Cuando llegas acá, es todo el proceso de RT Cores
OPTIX_INTERSECT_PROGRAM(Spheres)()
{

    // El optixGetPrimitiveIndex() devuelve el rayo que está intersectando en ese momento
    // getProgramData() devuelve el conjunto de datos principal del programa, en este caso el conjunto de esferas
	const int primID = optixGetPrimitiveIndex();
	const SpheresGeom &selfs = owl::getProgramData<SpheresGeom>();

    // extraigo la esfera ctual (Shader Core)
	Sphere self = selfs.data_pts[primID];

    // Te deuelve el punto donde se lanzó el rayo, osea la query
	const vec3f org = optixGetWorldRayOrigin();

    // Todo lo de arriba el Shader Core, sacamos los atributos que necesitamos, los que ocnseguimos con ayuda del BVH
    // ahora solo calcularemos los mejores vecinos.

	float distance = 0.0;

    // Acá definimos la norma a usar.
//#define NORM 0
//#define KN 20


    // Función de distancia que se va a usar en la NORM L^p
    // entre el centro de la geometría y la query.

#if (NORM == 0) // ceviche
	double x  = std::abs(self.center.x - org.x);
	double y = std::abs(self.center.y - org.y);
	double z = abs(self.center.z - org.z);
	if(x > y )
		distance = x;
	else
		distance = y;
	if(distance < z)
		distance = z;
#elif (NORM > 0) // p >= 1 pero no ceviche

    // no sacamos raiz, AÚN
  	distance = std::pow(std::abs(self.center.x - org.x), NORM)
			 + std::pow(std::abs(self.center.y - org.y), NORM)
			 + std::pow(std::abs(self.center.z - org.z), NORM);
#endif

      // Esta es la parte de filtrado, todos los que entren acá son posibles candidatos.
      // Acá se eleva al cuadrado porque estamos en el supuesto de que estamos usando norma euclidiana.
      // sería lo equivalente a poner srqt(distance, NORM) < self.rad
	if(distance < pow(selfs.rad, NORM)){

        NeighGroup &param = owl::getPRD<NeighGroup>();

        // Acá simplemente rescatamos el peor vecino, para comparar con el nuevo vecino actual
        // y si ver si es mejor que el peor vecino registrado.
        int max_idx=0;
        for (int i = 1; i < KN; i++){
          if (param.res[i].dist > param.res[max_idx].dist)
            max_idx = i;
        }

        // Parte de refinamiento, si cumple, refinamos los vecinos que ya teniamos.
        if ( distance < param.res[max_idx].dist) {

            // En el max_idx, guarda el peor vecino, acá guardamos el vecino actual, que es mejor que el peor.
            // lo guardamos tanto el indice DEL BLOQUE como tambien su distancia.
          param.res[max_idx].dist = distance;
          param.res[max_idx].ind = primID;
        }
  }

}


// Esto se activa internamente cuando se llama a owlLaunch2D
OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    // Rescatamos el rayo de la query
  const RayGenData &self = owl::getProgramData<RayGenData>();

  // Esto es fundamental para definir las mallas, en la lanzada de rayos en dif dimensiones.
  // SERÍA LOS puntos de consulta, lo que va a procesar el GPU en el lanzado de rayos.
  int xID = optixGetLaunchIndex().x;

  // Inicializo los vecinos con un valor incial.
  NeighGroup param;
  for(int i=0; i<KN; i++){
    param.res[i].ind = -1;
    param.res[i].dist = FLOAT_MAX;
  }

  // configuro el rayo a lanzar, con los datos de la query
  owl::Ray ray(self.query_pts[xID].center, vec3f(0,0,1), 0, 1.e-16f);

  // trazo el rayo en el escenario creado, guardando los vecinos en param.
  // esto llama a OPTIX_INTERSECT_PROGRAM, e
  owl::traceRay(self.world, ray, param);

  for(int i=0; i<KN; i++){
      // Esto guarda los resultados en la variable global, básicamente hace lo siguiente:
      // en base al punto de procesamiento xID, pues multiplixa con Kn + i, para decir
      //  xID = 0 [0, 19] xID = 1 [20, 39] xID= 2 [40, 59] ...
    optixLaunchParams.frameBuffer[xID*KN+i].ind = param.res[i].ind;
    optixLaunchParams.frameBuffer[xID*KN+i].dist = param.res[i].dist;
  }
}
